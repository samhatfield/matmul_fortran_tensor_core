
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

// Handles CUDA errors
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

// Handles cuBLAS errors
#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

// Converts from double-precision to half-precision (CUDA kernel)
__global__ void double2half(half *out, const double *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __float2half((float)(in[idx]));
    }
}

// Performs matrix-matrix multiplication using Tensor Core.
extern "C" {
    void tcgemm_c(int transa, int transb, int m, int n, int k, float alpha, void *a_p, int lda, void *b_p,
                int ldb, float beta, void *c_p, int ldc) {

        // Set up host-side arrays
        double *a_h, *b_h;
        float *c_h;
        a_h = (double *)a_p;
        b_h = (double *)b_p;
        c_h = (float *)c_p;

        // =========================================================================
        // Compute GEMM using Tensor Core
        // =========================================================================

        // Set up GPU and cuBLAS
        hipblasHandle_t cublasHandle;
        hipSetDevice(0);
        hipDeviceReset();
        cublasErrCheck(hipblasCreate(&cublasHandle));

        // Set up device-side arrays
        double *a_d, *b_d;
        half *a_d_16, *b_d_16;
        float *c_d_32;

        // Allocate memory on device for all arrays
        // TODO: should the dimensions used below (m*k etc.) take into account transa, lda etc.?
        cudaErrCheck(hipMalloc((void **)&a_d, m*k*sizeof(double)));
        cudaErrCheck(hipMalloc((void **)&b_d, k*n*sizeof(double)));
        cudaErrCheck(hipMalloc((void**)&a_d_16, m*k*sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_d_16, k*n*sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_d_32, m*n*sizeof(float)));

        // Copy input arrays to device
        cudaErrCheck(hipMemcpy(a_d, a_h, m*k*sizeof(double), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(b_d, b_h, k*n*sizeof(double), hipMemcpyHostToDevice));

        // Convert arrays to half-precision
        double2half<<<(int)((m*k)/256) + 1, 256 >>>(a_d_16, a_d, m*k);
        double2half<<<(int)((k*n)/256) + 1, 256 >>>(b_d_16, b_d, k*n);

        hipDeviceSynchronize();

        // Perform GEMM with Tensor Core
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
        cublasErrCheck(
                hipblasGemmEx(
                        cublasHandle, (hipblasOperation_t)transa, (hipblasOperation_t)transb,
                        m, n, k,
                        &alpha,
                        a_d_16, HIP_R_16F, lda,
                        b_d_16, HIP_R_16F, ldb,
                        &beta,
                        c_d_32, HIP_R_32F, ldc,
                        HIP_R_32F,
                        CUBLAS_GEMM_DEFAULT_TENSOR_OP
                )
        );

        // Copy results back from device to host
        cudaErrCheck(hipMemcpy(c_h, c_d_32, m*n*sizeof(float), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();

        // Free memory on device
        cudaErrCheck(hipFree(a_d));
        cudaErrCheck(hipFree(b_d));
        cudaErrCheck(hipFree(a_d_16));
        cudaErrCheck(hipFree(b_d_16));
        cudaErrCheck(hipFree(c_d_32));
    }
}
