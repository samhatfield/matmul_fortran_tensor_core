
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

// Handles CUDA errors
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

// Handles cuBLAS errors
#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

// Performs matrix-matrix multiplication using Tensor Core.
extern "C" {
    void tcgemm_c(char transa, char transb, int m, int n, int k, double alpha, void *a_p, int lda, void *b_p,
                int ldb, double beta, void *c_p, int ldc) {
    
        // Set up host-side arrays
        double *a_h, *b_h, *c_h;
        a_h = (double *)a_p;
        b_h = (double *)b_p;
        c_h = (double *)c_p;
    
        // =========================================================================
        // Compute GEMM using Tensor Core
        // =========================================================================
    
        // Set up GPU and cuBLAS
        hipblasHandle_t cublasHandle;
        hipSetDevice(0);
        hipDeviceReset();
        cublasErrCheck(hipblasCreate(&cublasHandle));
    
        // Set up device-side arrays
        double *a_d, *b_d, *c_d;
    
        // Allocate memory on device for all arrays
        // TODO: should the dimensions used below (m*k etc.) take into account transa, lda etc.?
        cudaErrCheck(hipMalloc((void **)&a_d, m*k*sizeof(double)));
        cudaErrCheck(hipMalloc((void **)&b_d, k*n*sizeof(double)));
        cudaErrCheck(hipMalloc((void **)&c_d, m*n*sizeof(double)));
    
        // Copy input arrays to device
        cudaErrCheck(hipMemcpy(a_d, a_h, m*k*sizeof(double), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(b_d, b_h, k*n*sizeof(double), hipMemcpyHostToDevice));
    
        hipblasOperation_t transa_int = (transa == 'N' || transa == 'n') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
        hipblasOperation_t transb_int = (transb == 'N' || transb == 'n') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    
        // Perform GEMM
        cublasErrCheck(
                hipblasGemmEx(
                        cublasHandle, transa_int, transb_int,
                        m, n, k,
                        &alpha,
                        a_d, HIP_R_64F, lda,
                        b_d, HIP_R_64F, ldb,
                        &beta,
                        c_d, HIP_R_64F, ldc,
                        HIP_R_64F,
                        HIPBLAS_GEMM_DEFAULT
                )
        );
    
        // Copy results back from device to host
        cudaErrCheck(hipMemcpy(c_h, c_d, m*n*sizeof(double), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();

        // Free memory on device
        cudaErrCheck(hipFree(a_d));
        cudaErrCheck(hipFree(b_d));
        cudaErrCheck(hipFree(c_d));
    
        // =========================================================================
    
        // Set incoming C array pointer
        //c_p = (void *)c_h;
    }
}
